#include "hip/hip_runtime.h"
#include <stdio.h>
#include <cv.h>
#include <hip/hip_runtime.h>

using namespace std;

#define BLUE 0
#define GREEN 1
#define RED 2
#define B_WEIGHT 0.1140
#define G_WEIGHT 0.5870
#define R_WEIGHT 0.2989

void checkCudaError(hipError_t err) {
  if (err != hipSuccess) {
    printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
    exit(EXIT_FAILURE);
  }
}

__global__ ImageToGrayscale(unsigned char *d_img, unsigned char *d_gray, int rows, int cols) {
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  if (row < rows && col < cols) {
    int i = row * cols + col;
    d_gray[i] = B_WEIGHT * img[i * 3 + BLUE] +
                G_WEIGHT * img[i * 3 + GREEN] +
                R_WEIGHT * img[i * 3 + RED];
  }
}

SeqImageToGrayscale(unsigned char *h_img, unsigned char *h_gray, int rows, int cols) {
  for (int i = 0; i < rows*cols; i++) {
    h_gray[i] = B_WEIGHT * h_img[i * 3 + BLUE] +
                G_WEIGHT * h_img[i * 3 + GREEN] +
                R_WEIGHT * h_img[i * 3 + RED];
  }
}

int main(int argc, char *argv[]) {
  if (argc != 2) {
    cout << "Usage: img2grayscale.out <image_name>" << endl;
    return -1;
  }
  clock_t startCPU, endCPU, startGPU, endGPU;
  double cpu_time_used, gpu_time_used;

  Mat img = imread(argv[1]);

  if (!img.data) {
    cout << "Error reading image, it may not exist" << endl;
  }

  int width = img.size().width;
  int height = img.size().height;
  int N = width * height;
  size_t num_bytes = N * 3 * sizeof(unsigned char);

  Mat gray_gpu(width, height, img.type());
  Mat gray_cpu(width, height, img.type());

  unsigned char *h_img = img.data;
  unsigned char *h_gray_img = (unsigned char *) malloc(num_bytes);
  unsigned char *gray_img = (unsigned char *) malloc(num_bytes);
  unsigned char *d_img;
  unsigned char *d_gray_img;

  hipError_t err;
  err = hipMalloc((void **) &d_img, num_bytes);
  checkCudaError(err);
  err = hipMalloc((void **) &d_gray_img, num_bytes);
  checkCudaError(err);

  // Image to grayscale in GPU
  startGPU = clock();

  err = hipMemcpy(d_img, h_img, num_bytes, hipMemcpyHostToDevice);
  checkCudaError();

  int block_size = 32;
  dim3 block_dim(block_size, block_size, 1);
  dim3 grid_dim(ceil(width / float(block_size)), ceil(height / float(block_size)), 1);

  ImgToGrayscale<<<grid_dim, block_dim>>>(d_img, d_gray_img, width, height);
  hipDeviceSynchronize();

  err = hipMemcpy(h_gray_img, d_gray_img, num_bytes, hipMemcpyDeviceToHost);
  checkCudaError();

  endGPU = clock();

  // Image to grayscale in CPU
  startCPU = clock();

  SeqImgToGrayscale(h_img, gray_img, width, height);

  endCPU = clock();

  // Generating result images
  gray_gpu.data = h_gray_img;
  gray_cpu.data = gray_img;

  // Show results
  namedWindow("Image", WINDOW_NORMAL);
  namedWindow("Image to grayscale in GPU", WINDOW_NORMAL);
  namedWindow("Image to grayscale in CPU", WINDOW_NORMAL);
  imshow("Image", img);
  imshow("Image to grayscale in CPU", gray_cpu);
  imshow("Image to grayscale in GPU", gray_gpu);
  waitKey(0);

  // Generating times
  gpu_time_used = ((double) (endGPU - startGPU)) / CLOCKS_PER_SEC;
  printf("Time in GPU: %.10f\n",gpu_time_used);
  cpu_time_used = ((double) (endCPU - startCPU)) / CLOCKS_PER_SEC;
  printf("Time in CPU: %.10f\n",cpu_time_used);
  printf("Acceleration: %.10fX\n", cpu_time_used / gpu_time_used);

  free(h_gray_img_result); free(gray_img_result);
  hipFree(d_img); hipFree(d_gray_img_result);
  return 0;
}
