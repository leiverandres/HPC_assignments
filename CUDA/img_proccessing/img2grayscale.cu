#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>

using namespace std;
using namespace cv;

__global__ void matByConst(unsigned char *img, int alpha, int N) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < N) {
    img[i] *= 2;
  }
}

int main(int argc, char *argv[]) {
  if (argc != 2) {
    cout << "Usage: img2grayscale.out <image_name>" << endl;
    return -1;
  }

  Mat img = imread(argv[1], CV_LOAD_IMAGE_GRAYSCALE);
  Mat result(img.size(), CV_8UC1);
  unsigned char *h_img = (unsigned char *) img.data;
  unsigned char *d_img;
  
  int N = img.rows * img.cols;
  size_t num_bytes = N * sizeof(unsigned char);
  int block_size = 256;
  int blocks = ceil(N / block_size);
 
  hipMalloc((void **) &d_img, num_bytes);
  hipMemcpy(d_img, h_img, num_bytes, hipMemcpyHostToDevice);

  matByConst<<<blocks, block_size>>>(d_img, 2, N);
  
  hipError_t err = hipMemcpy((unsigned char *) &result.data, d_img, num_bytes, hipMemcpyDeviceToHost);
  
  if (err != hipSuccess) {
    printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
    exit(EXIT_FAILURE);
  } 
  imshow("Image in grayscale", img);
  waitKey(0);
  imshow("Image after multiplication", result);
  hipFree(d_img);
  return 0;
}

