#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <iostream>

using namespace std;

void checkCudaError(hipError_t err) {
  if (err != hipSuccess) {
    printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
    exit(EXIT_FAILURE);
  }
}

__global__ 
void Multiplication(float *A, float *B, float *C, int A_ROWS, int A_COLS, int B_COLS) {
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  if (row < A_ROWS && col < B_COLS) {
    float sum = 0.0;
    for (int i = 0; i < A_COLS; i++) {
      sum += A[row * A_COLS + i] * B[i * B_COLS + col];
    }
    C[row * B_COLS + col] = sum;
  }
}

void SeqMultiplication(float *A, float *B, float *C, int A_ROWS, int A_COLS, int B_COLS) {
  for (int i = 0; i < A_ROWS; i++) {
    for (int j = 0; j < B_COLS; j++) {
      float sum = 0.0;
      for (int k = 0; k < A_COLS; k++) {
        sum += A[i * A_COLS + k] * B[k * B_COLS + j]; 
      }
      C[i * B_COLS + j] = sum;
    }
  }
}

void InitMat(float *mat, int rows, int cols) {
  for (int i = 0; i < rows * cols; i++) {
    mat[i] = rand() % 50;
  }
}

void ShowMat(string name, float *mat, int rows, int cols) {
  cout << name << " = [";
  for (int i = 0; i < rows; i++) {
    for (int j = 0; j < cols; j++) {
      cout << mat[i * cols + j] << " ";
    }
    cout << ";" << endl;
  }
  cout << "]" << endl;
}

bool CheckResults(float *x, float *y, int rows, int cols) {
  for (int i = 0; i < rows; i++) {
    for (int j = 0; j < cols; j++) {
      if (x[i * cols + j] != y[i * cols + j]) {
        return false;
      }
    }
  }
  return true;
}

void RunTests(int num_test) {
  for (int i = 0; i < num_test; i++) {
    
  }
}

void PrintUsage(string program) {
  cout << "Usage: " << program << " A_ROWS A_COLS B_COLS [OPTIONS]" << endl;
  cout << "* Is not needed to pass B_ROWS because B_ROWS must be equal to A_COLS";
  cout << endl << "OPTIONS: " << endl;
  cout << "-p: Print matrices" << endl;
  cout << endl;
}

int main(int argc, char *argv[]) {
  int A_ROWS, A_COLS, B_ROWS, B_COLS;
  if (argc < 4) {
    PrintUsage(argv[0]);
    return -1;
  } else {
    A_ROWS = atoi(argv[1]);
    A_COLS = atoi(argv[2]);
    B_ROWS = A_COLS;
    B_COLS = atoi(argv[3]);
  }
  clock_t startCPU, endCPU, startGPU, endGPU;
  double cpu_time_used, gpu_time_used;
  
  size_t size_a = A_ROWS * A_COLS * sizeof(float);
  size_t size_b = B_ROWS * B_COLS * sizeof(float);
  size_t size_c = A_ROWS * B_COLS * sizeof(float);

  float *h_a = (float *) malloc(size_a);
  float *h_b = (float *) malloc(size_b);
  float *h_c = (float *) malloc(size_c);
  float *d_a;
  float *d_b;
  float *d_c;
  float *seq_c = (float *) malloc(size_c);

  // Init matrices
  InitMat(h_a, A_ROWS, A_COLS);
  InitMat(h_b, B_ROWS, B_COLS);

  hipError_t err;
  err = hipMalloc((void **) &d_a, size_a);
  checkCudaError(err);
  err = hipMalloc((void **) &d_b, size_b);
  checkCudaError(err);
  err = hipMalloc((void **) &d_c, size_c);
  checkCudaError(err);

  // Multiplication in GPU
  startGPU = clock();
  
  err = hipMemcpy(d_a, h_a, size_a, hipMemcpyHostToDevice);
  checkCudaError(err);
  err = hipMemcpy(d_b, h_b, size_b, hipMemcpyHostToDevice);
  checkCudaError(err);

  int block_size = 32;
  dim3 block_dim(block_size, block_size, 1);
  dim3 grid_dim(ceil((float)B_COLS / block_size), ceil((float)A_ROWS / block_size), 1);

  Multiplication<<<grid_dim, block_dim>>>(d_a, d_b, d_c, A_ROWS, A_COLS, B_COLS);
  hipDeviceSynchronize();

  err = hipMemcpy(h_c, d_c, size_c, hipMemcpyDeviceToHost);
  checkCudaError(err);

  endGPU = clock();

  // Image to grayscale in CPU
  startCPU = clock();

  SeqMultiplication(h_a, h_b, seq_c, A_ROWS, A_COLS, B_COLS);

  endCPU = clock();
  
  // Show matriced
  if (argv[4]) {
    ShowMat("A", h_a, A_ROWS, A_COLS);
    ShowMat("B", h_b, B_ROWS, B_COLS);
    ShowMat("S_C", seq_c, A_ROWS, B_COLS);
    ShowMat("P_C", h_c, A_ROWS, B_COLS);    
  }
 
  // Generating times
  if (CheckResults(h_c, seq_c, A_ROWS, B_COLS)) {
    gpu_time_used = ((double) (endGPU - startGPU)) / CLOCKS_PER_SEC;
    printf("Time in GPU: %.10f\n",gpu_time_used);
    cpu_time_used = ((double) (endCPU - startCPU)) / CLOCKS_PER_SEC;
    printf("Time in CPU: %.10f\n",cpu_time_used);
    printf("Acceleration: %.10fX\n", cpu_time_used / gpu_time_used);
  } else {
    printf("Results are not consistent\n");
  }

  free(h_a); free(h_b); free(h_c); free(seq_c);
  hipFree(d_a); hipFree(d_b); hipFree(d_c);
  return 0;
}
