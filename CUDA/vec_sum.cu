#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <time.h>
#define N 1024

void seqSum(int *a, int *b, int *c, int size) {
  for (int i = 0; i < size; i++)
    c[i] = a[i] + b[i];
}

void fillRandomVec(int *x, int size) {
  for (int i = 0; i < size; i++)
    x[i] = rand() % 50;
}

void showVec(int *x, int size) {
  for (int i = 0; i < size; i++)
    printf("%d ", x[i]);
  printf("\n");
}

void compareResults(int *ans1, int *ans2, int size) {
  for (int i = 0; i < size; i++) {
    if (ans1[i] != ans2[i]) {
       printf("Sum comparison failed at %d index\n", i);
       return;
    }
  }
  printf("Answers are the same");
}

__global__ void deviceAddVector(int *d_a, int *d_b, int *d_c, int size) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < size) {
    d_c[i] = d_a[i] + d_b[i];
    //  printf("Tread %d make sum %d + %d = %d", i, d_a[i], d_b[i], d_c[i]);
  }
}

void checkCudaError(hipError_t err) {
  if (err != hipSuccess) {
    printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
    exit(EXIT_FAILURE);
  } 
}

int main(int argc, char *argv[]) {
  int *h_a, *h_b, *h_c1, *h_c2;
  int *d_a, *d_b, *d_c;
  size_t num_bytes = N * sizeof(int);
  
  h_a = (int *) malloc(num_bytes);
  h_b = (int *) malloc(num_bytes);
  h_c1 = (int *) malloc(num_bytes);
  h_c2 = (int *) malloc(num_bytes);
  
  fillRandomVec(h_a, N);
  fillRandomVec(h_b, N);
  
  hipError_t err_a = hipMalloc((void **) &d_a, num_bytes);
  hipError_t err_b = hipMalloc((void **) &d_b, num_bytes);
  hipError_t err_c = hipMalloc((void **) &d_c, num_bytes);  
  checkCudaError(err_a);
  checkCudaError(err_b);
  checkCudaError(err_c);

  hipError_t err_cpy_a = hipMemcpy(d_a, h_a, num_bytes, hipMemcpyHostToDevice);
  hipError_t err_cpy_b = hipMemcpy(d_b, h_b, num_bytes, hipMemcpyHostToDevice);

  checkCudaError(err_cpy_a);
  checkCudaError(err_cpy_b);

  seqSum(h_a, h_b, h_c1, N);

  // showVec(h_a, N); showVec(h_b, N); showVec(h_c1, N);

  int block_size = min(256, N);
  int num_blocks = ceil(N / block_size);
  printf("%d blocks, %d per threads per block\n", num_blocks, block_size);
  deviceAddVector<<<num_blocks, block_size>>>(d_a, d_b, d_c, N);

  hipError_t err_cpy_c = hipMemcpy(h_c2, d_c, num_bytes, hipMemcpyDeviceToHost);
  
  checkCudaError(err_cpy_c);
  // showVec(h_c2, N);
  compareResults(h_c1, h_c2, N);
  
  free(h_a); free(h_b); free(h_c1); free(h_c2);
  hipFree(d_a); hipFree(d_b); hipFree(d_c);
  return 0;
}
