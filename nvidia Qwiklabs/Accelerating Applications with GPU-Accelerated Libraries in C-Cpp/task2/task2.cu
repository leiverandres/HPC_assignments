#include "hip/hip_runtime.h"
#include "hipblas.h"
#include "hiprand.h"
#include "timer.h"

#include <stdio.h>

#define INDX( row, col, ld ) ( ( (col) * (ld) ) + (row) )

#define SIZE 10000

int main()
{
    const int size = SIZE;

    fprintf(stdout, "Matrix size is %d\n",size);

    float *h_a, *h_b, *h_c;
    float *d_a, *d_b, *d_c;
 
    size_t numbytes = size * size * sizeof( float );

    // Allocate all our host-side (CPU) data
    h_a = (float *) malloc( numbytes );
    if( h_a == NULL )
    {
      fprintf(stderr,"Error in host malloc\n");
      return 911;
    }

    h_b = (float *) malloc( numbytes );
    if( h_b == NULL )
    {
      fprintf(stderr,"Error in host malloc\n");
      return 911;
    }

    h_c = (float *) malloc( numbytes );
    if( h_c == NULL )
    {
      fprintf(stderr,"Error in host malloc\n");
      return 911;
    }

    hipblasHandle_t handle;
    hipblasStatus_t stat = hipblasCreate( &handle );

    // Set these constants so we get a simple matrix multiply with hipblasSgemm
    float alpha = 1.0;
    float beta  = 0.0;
  
    StartTimer();
  
    // Allocate device-side (GPU) memory
    hipMalloc( (void **)&d_a, numbytes );
    hipMalloc( (void **)&d_b, numbytes );
    hipMalloc( (void **)&d_c, numbytes );

    // Generate size * size random numbers
    printf("Create random numbers\n");
    // FIXME: Replace the following for-loop with two hiprandGenerateNormal calls

    hiprandGenerator_t gen;
    hiprandCreateGeneratorHost(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandGenerateNormal(gen, h_a, size*size, 0.0, float(RAND_MAX));    
    hiprandGenerateNormal(gen, h_b, size*size, 0.0, float(RAND_MAX));


  	// Copy the a and b matrices to the GPU memory using hipMemcpy
    hipMemcpy( d_a, h_a, numbytes, hipMemcpyHostToDevice );
    hipMemcpy( d_b, h_b, numbytes, hipMemcpyHostToDevice );

    // Launch hipblasSgemm on the GPU
    printf("Launching GPU sgemm\n");
    hipblasSgemm( handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                 size, size, size,
                 &alpha, 
                 d_a, size,
                 d_b, size,
                 &beta,
                 d_c, size );

    // Finally, copy the resulting c array back to the host  
    hipMemcpy( h_c, d_c, numbytes, hipMemcpyDeviceToHost );

    double runtime = GetTimer();

    fprintf(stdout, "Total time is %f sec\n", runtime / 1000.0f );

    hipblasDestroy( handle );
	hiprandDestroyGenerator(gen);	
  
    hipFree( d_a );
    hipFree( d_b );
    hipFree( d_c );

    free( h_a );
    free( h_b );
    free( h_c );

    return 0;
}
